#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cmath>
#include <cstdint>
#include <stdexcept>
#include <string>
#include <vector>

#include "bitboard.h"
#include "board.h"
#include "nnue/network.h"
#include "training/gpu_backend.h"

namespace chiron::gpu {

namespace {

void check_cuda(hipError_t result, const char* context) {
    if (result != hipSuccess) {
        throw std::runtime_error(std::string("CUDA error during ") + context + ": " +
                                 hipGetErrorString(result));
    }
}

__device__ inline int clamp_weight_device(double value) {
    double rounded = nearbyint(value);
    if (rounded > static_cast<double>(kTrainerWeightLimit)) {
        rounded = static_cast<double>(kTrainerWeightLimit);
    }
    if (rounded < static_cast<double>(-kTrainerWeightLimit)) {
        rounded = static_cast<double>(-kTrainerWeightLimit);
    }
    return static_cast<int>(rounded);
}

__global__ void train_example_kernel(const int8_t* features, int target_cp, int orientation,
                                     double learning_rate, double regularisation, int hidden_size,
                                     int feature_count, int32_t* input_weights, int32_t* hidden_biases,
                                     float* output_weights, int32_t* bias, float scale) {
    extern __shared__ double shared[];
    double* activations = shared;
    double* derivatives = activations + hidden_size;
    double* lr_error_storage = derivatives + hidden_size;

    int tid = threadIdx.x;
    if (tid < hidden_size) {
        long long offset = static_cast<long long>(tid) * feature_count;
        double pre = static_cast<double>(hidden_biases[tid]);
        for (int f = 0; f < feature_count; ++f) {
            int8_t feature = features[f];
            if (feature == 0) {
                continue;
            }
            pre += static_cast<double>(input_weights[offset + f]) * static_cast<double>(feature);
        }
        double normalized = pre / nnue::kActivationScale;
        double tanh_val = tanh(normalized);
        activations[tid] = tanh_val * nnue::kActivationScale;
        derivatives[tid] = 1.0 - tanh_val * tanh_val;
    }
    __syncthreads();

    if (tid == 0) {
        double raw = static_cast<double>(*bias);
        for (int j = 0; j < hidden_size; ++j) {
            raw += activations[j] * static_cast<double>(output_weights[j]);
        }
        double predicted_cp = static_cast<double>(orientation) * raw * static_cast<double>(scale);
        double error = static_cast<double>(target_cp) - predicted_cp;
        double lr_error = learning_rate * error * static_cast<double>(orientation) * static_cast<double>(scale);
        lr_error_storage[0] = lr_error;

        double bias_current = static_cast<double>(*bias);
        double bias_next = bias_current + lr_error;
        if (regularisation > 0.0) {
            bias_next -= regularisation * bias_current;
        }
        *bias = clamp_weight_device(bias_next);
    }
    __syncthreads();

    if (tid >= hidden_size) {
        return;
    }

    double lr_error = lr_error_storage[0];
    double activation = activations[tid];
    double output_current = static_cast<double>(output_weights[tid]);
    double output_next = output_current + lr_error * activation;
    if (regularisation > 0.0) {
        output_next -= regularisation * output_current;
    }
    output_weights[tid] = static_cast<float>(output_next);

    double grad_pre = lr_error * output_current * derivatives[tid];
    double hidden_current = static_cast<double>(hidden_biases[tid]);
    double hidden_next = hidden_current + grad_pre;
    if (regularisation > 0.0) {
        hidden_next -= regularisation * hidden_current;
    }
    hidden_biases[tid] = clamp_weight_device(hidden_next);

    if (fabs(grad_pre) < 1e-12) {
        return;
    }

    long long offset = static_cast<long long>(tid) * feature_count;
    for (int f = 0; f < feature_count; ++f) {
        int8_t feature = features[f];
        if (feature == 0) {
            continue;
        }
        int32_t current = input_weights[offset + f];
        double next = static_cast<double>(current) + grad_pre * static_cast<double>(feature);
        if (regularisation > 0.0) {
            next -= regularisation * static_cast<double>(current);
        }
        input_weights[offset + f] = clamp_weight_device(next);
    }
}

void encode_features(const Board& board, std::vector<int8_t>& buffer) {
    std::fill(buffer.begin(), buffer.end(), 0);
    for (int color = 0; color < kNumColors; ++color) {
        for (int piece = 0; piece < kNumPieceTypes; ++piece) {
            Bitboard bb = board.pieces(static_cast<Color>(color), static_cast<PieceType>(piece));
            while (bb) {
                int square = pop_lsb(bb);
                std::size_t feature =
                    nnue::feature_index(static_cast<Color>(color), static_cast<PieceType>(piece), square);
                buffer[feature] = (color == static_cast<int>(Color::White)) ? 1 : -1;
            }
        }
    }
}

}  // namespace

void train_batch_cuda(const std::vector<TrainingExample>& batch, nnue::Network& network,
                      const Trainer::Config& config) {
    if (batch.empty()) {
        return;
    }

    int hidden = static_cast<int>(network.hidden_size());
    if (hidden <= 0) {
        return;
    }
    int feature_count = static_cast<int>(nnue::kFeatureCount);

    auto& input_weights = network.input_weights_data();
    auto& hidden_biases = network.hidden_biases_data();
    auto& output_weights = network.output_weights_data();
    int32_t bias_value = network.bias();
    float scale_value = network.scale();

    int32_t* d_input_weights = nullptr;
    int32_t* d_hidden_biases = nullptr;
    float* d_output_weights = nullptr;
    int32_t* d_bias = nullptr;
    int8_t* d_features = nullptr;

    try {
        check_cuda(hipMalloc(&d_input_weights, input_weights.size() * sizeof(int32_t)), "hipMalloc input weights");
        check_cuda(hipMalloc(&d_hidden_biases, hidden_biases.size() * sizeof(int32_t)),
                   "hipMalloc hidden biases");
        check_cuda(hipMalloc(&d_output_weights, output_weights.size() * sizeof(float)),
                   "hipMalloc output weights");
        check_cuda(hipMalloc(&d_bias, sizeof(int32_t)), "hipMalloc bias");
        check_cuda(hipMalloc(&d_features, static_cast<size_t>(feature_count) * sizeof(int8_t)),
                   "hipMalloc features");

        check_cuda(hipMemcpy(d_input_weights, input_weights.data(),
                              input_weights.size() * sizeof(int32_t), hipMemcpyHostToDevice),
                   "hipMemcpy input weights to device");
        check_cuda(hipMemcpy(d_hidden_biases, hidden_biases.data(),
                              hidden_biases.size() * sizeof(int32_t), hipMemcpyHostToDevice),
                   "hipMemcpy hidden biases to device");
        check_cuda(hipMemcpy(d_output_weights, output_weights.data(),
                              output_weights.size() * sizeof(float), hipMemcpyHostToDevice),
                   "hipMemcpy output weights to device");
        check_cuda(hipMemcpy(d_bias, &bias_value, sizeof(int32_t), hipMemcpyHostToDevice),
                   "hipMemcpy bias to device");

        std::vector<int8_t> feature_buffer(static_cast<std::size_t>(feature_count), 0);

        for (const TrainingExample& example : batch) {
            Board board;
            board.set_from_fen(example.fen);
            encode_features(board, feature_buffer);

            check_cuda(hipMemcpy(d_features, feature_buffer.data(),
                                  static_cast<size_t>(feature_count) * sizeof(int8_t), hipMemcpyHostToDevice),
                       "hipMemcpy features to device");

            int orientation = board.side_to_move() == Color::White ? 1 : -1;
            int target = example.target_cp;
            int threads = 1;
            while (threads < hidden) {
                threads <<= 1;
            }
            if (threads > 1024) {
                threads = 1024;
            }
            std::size_t shared_bytes = static_cast<std::size_t>(2 * hidden + 1) * sizeof(double);

            train_example_kernel<<<1, threads, shared_bytes>>>(d_features, target, orientation,
                                                               config.learning_rate, config.regularisation, hidden,
                                                               feature_count, d_input_weights, d_hidden_biases,
                                                               d_output_weights, d_bias, scale_value);
            check_cuda(hipGetLastError(), "launch train_example_kernel");
            check_cuda(hipDeviceSynchronize(), "train_example_kernel");
        }

        check_cuda(hipMemcpy(input_weights.data(), d_input_weights,
                              input_weights.size() * sizeof(int32_t), hipMemcpyDeviceToHost),
                   "hipMemcpy input weights to host");
        check_cuda(hipMemcpy(hidden_biases.data(), d_hidden_biases,
                              hidden_biases.size() * sizeof(int32_t), hipMemcpyDeviceToHost),
                   "hipMemcpy hidden biases to host");
        check_cuda(hipMemcpy(output_weights.data(), d_output_weights,
                              output_weights.size() * sizeof(float), hipMemcpyDeviceToHost),
                   "hipMemcpy output weights to host");
        check_cuda(hipMemcpy(&bias_value, d_bias, sizeof(int32_t), hipMemcpyDeviceToHost),
                   "hipMemcpy bias to host");

        network.set_bias(bias_value);
    } catch (...) {
        if (d_features) hipFree(d_features);
        if (d_bias) hipFree(d_bias);
        if (d_output_weights) hipFree(d_output_weights);
        if (d_hidden_biases) hipFree(d_hidden_biases);
        if (d_input_weights) hipFree(d_input_weights);
        throw;
    }

    if (d_features) hipFree(d_features);
    if (d_bias) hipFree(d_bias);
    if (d_output_weights) hipFree(d_output_weights);
    if (d_hidden_biases) hipFree(d_hidden_biases);
    if (d_input_weights) hipFree(d_input_weights);
}

}  // namespace chiron::gpu
